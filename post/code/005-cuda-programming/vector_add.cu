#include <iostream>
#include <hip/hip_runtime.h>

// __global__ indicates the add function is a kernel that runs on the GPU
__global__ void add(int *a, int *b, int *c, int size)
{
    // built-in variables define the index of the block and thread. 
    // Cuda organizesd execution in grids of blocks, and each block contains multiple threads.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

#define cudaCheckError() {                                                    \
    hipError_t e = hipGetLastError();                                       \
    if (e != hipSuccess) {                                                   \
        printf("CUDA Error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                                                   \
    }                                                                         \
}

int main()
{
    const int size = 5;

    int h_a[size] = {1, 2, 3, 4, 5};
    int h_b[size] = {10, 20, 30, 40, 50};
    int h_c[size] = {0};

    int *d_a, *d_b, *d_c;
    size_t bytes = size * sizeof(int);

    // Allocate memory on the GPU
    hipMalloc(&d_a, bytes);
    cudaCheckError();
    hipMalloc(&d_b, bytes);
    cudaCheckError();
    hipMalloc(&d_c, bytes);
    cudaCheckError();

    // Copy data from host to device (CPU -> GPU)
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    cudaCheckError();

    // Launch kernel with 1 block of 5 threads
    add<<<1, size>>>(d_a, d_b, d_c, size);
    cudaCheckError();

    // Copy the result back to the host (GPU -> CPU)
    
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    cudaCheckError();

    // Print the result
    std::cout << "Result: ";
    for (int i = 0; i < size; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}